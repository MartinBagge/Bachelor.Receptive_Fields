#include "hip/hip_runtime.h"
/*
 * parallelize.cpp
 *
 *  Created on: Apr 24, 2016
 *      Author: martin
 */

#include "Parallelize.cuh"

Parallelize::Parallelize() {
	// TODO Auto-generated constructor stub

}

Parallelize::~Parallelize() {
	// TODO Auto-generated destructor stub
}

__global__ void d_createKernel(double *center, double *step, double *width, double *kernel, int size){
	int i = threadIdx.x;
	if(i < size)
		*kernel = exp((-(((*center)-(*step))*((*center)-(*step)))/2)*(*width));
}

void d_createKernels(double *centers, double step, double width, double *kernelsArr){
	  int N = sizeof(centers);
	  double stepArr[1], widthArr[1], *d_kernelCenter, *d_step, *d_kernelWidth, *d_kernel;

	  std::cout << "start malloc" << std::endl;

	  hipMalloc(&d_kernelCenter, N*sizeof(double));
	  hipMalloc(&d_step, N*sizeof(double));
	  hipMalloc(&d_kernelWidth, N*sizeof(double));
	  hipMalloc(&d_kernel, N*sizeof(double));

	  std::cout << "end malloc" << std::endl;


	  for(int i = 0; i < N; i++){
		  stepArr[i] = step;
		  widthArr[i] = width;
		  kernelsArr[i] = 0;
	  }

	  std::cout << "start memcpy" << std::endl;

	  hipMemcpy(d_kernelCenter, centers, N*sizeof(double), hipMemcpyHostToDevice);
	  hipMemcpy(d_step, stepArr, N*sizeof(double), hipMemcpyHostToDevice);
	  hipMemcpy(d_kernelWidth, widthArr, N*sizeof(double), hipMemcpyHostToDevice);
	  hipMemcpy(d_kernel, kernelsArr, N*sizeof(double), hipMemcpyHostToDevice);

	  std::cout << "end memcpy" << std::endl;

	  //function call
	  std::cout << "start call function" << std::endl;
	  d_createKernel <<<1, N>>>(d_kernelCenter, d_step, d_kernelWidth, d_kernel, N);

	  std::cout << "end function call" << std::endl;

	  std::cout << "start memcpy" << std::endl;

	  hipMemcpy(centers, d_kernelCenter, N*sizeof(double), hipMemcpyDeviceToHost);
	  hipMemcpy(stepArr, d_step, N*sizeof(double), hipMemcpyDeviceToHost);
	  hipMemcpy(widthArr, d_kernelWidth, N*sizeof(double), hipMemcpyDeviceToHost);
	  hipMemcpy(kernelsArr, d_kernel, N*sizeof(double), hipMemcpyDeviceToHost);

	  std::cout << "end memcpy" << std::endl;

	  std::cout << "start free" << std::endl;


	  hipFree(d_kernelCenter);
	  hipFree(d_step);
	  hipFree(d_kernelWidth);
	  hipFree(d_kernel);


}

std::vector<double> Parallelize::createKernels(std::vector<double> centers, double step, double width, int size){
	double* centersArr = &centers[0];
	double* kernelsArr = new double[size];
	d_createKernels(centersArr, step, width, kernelsArr);
	std::vector<double> returnVector;
	for(int i = 0; i < size; i++){
		returnVector.push_back(kernelsArr[i]);
	}
	delete[] centersArr;
	delete[] kernelsArr;
	return returnVector;
}
