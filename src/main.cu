#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <vector>
#include <fstream>
#include <sstream>
#include "ReceptiveFields.h"
#include <gtest/gtest.h>

using namespace std;
/*
TEST(Re){

}
*/

__global__ void createKernel(double *kernelCenter, double *step, double *kernelWidth, double *kernel, int n){
	int i = threadIdx.x;
	if(i < n)
		*kernel = exp((-(((*kernelCenter)-(*step))*((*kernelCenter)-(*step)))/2)*(*kernelWidth));
}

int main(){
	
	cout << "Test" << endl;

	ReceptiveFields RF(60, 120, 40, 0.1, 0.4, 50000, 85);
	//Read data file
	string inputString;
	string token;
	vector<string> splitStrings;
	ifstream filein;
	filein.open("runbot_leftknee_output.log");
	//while(getline(filein,inputString)){
			//stringstream lineStream(inputString);
			//splitStrings.clear();
	std::cout << "fileread" << std::endl;
			while(getline(filein, token, ',')){
				splitStrings.push_back(token);
			}
			for(int i = 0; i < splitStrings.size(); i++){
				RF.generateTarget(atof(splitStrings[i].c_str()));
			}
			std::cout << "fileread done" << std::endl;
		//}
	int tmp = 1;
	for(int i = 0; i < 85; i++){
		RF.createStep(tmp);
		tmp++;
	}
	RF.applyDeltaRule();
	/*
	cout << "start" << endl;

	int N = 1;
	  double kernelCenter[1], step[1], kernelWidth[1], kernel[1], *d_kernelCenter, *d_step, *d_kernelWidth, *d_kernel;
	  
	  cout << "start malloc" << endl;

	  hipMalloc(&d_kernelCenter, N*sizeof(double));
	  hipMalloc(&d_step, N*sizeof(double));
	  hipMalloc(&d_kernelWidth, N*sizeof(double));
	  hipMalloc(&d_kernel, N*sizeof(double));
	  
	  cout << "end malloc" << endl;
	  
	  kernelCenter[0] = 41.9231f;
	  step[0] = 20.0f;
	  kernelWidth[0] = 0.1f;
	  kernel[0] = 0;
	  
	  cout << "start memcpy" << endl;

	  hipMemcpy(d_kernelCenter, kernelCenter, N*sizeof(double), hipMemcpyHostToDevice);
	  hipMemcpy(d_step, step, N*sizeof(double), hipMemcpyHostToDevice);
	  hipMemcpy(d_kernelWidth, kernelWidth, N*sizeof(double), hipMemcpyHostToDevice);
	  hipMemcpy(d_kernel, kernel, N*sizeof(double), hipMemcpyHostToDevice);
	  
	  cout << "end memcpy" << endl;

	  //function call
	  cout << "start call function" << endl;
	  createKernel <<<1, N>>>(d_kernelCenter, d_step, d_kernelWidth, d_kernel, N);
	  cout << "end function call" << endl;
	  
	  cout << "start memcpy" << endl;

	  hipMemcpy(kernelCenter, d_kernelCenter, N*sizeof(double), hipMemcpyDeviceToHost);
	  hipMemcpy(step, d_step, N*sizeof(double), hipMemcpyDeviceToHost);
	  hipMemcpy(kernelWidth, d_kernelWidth, N*sizeof(double), hipMemcpyDeviceToHost);
	  hipMemcpy(kernel, d_kernel, N*sizeof(double), hipMemcpyDeviceToHost);
	  
	  cout << "end memcpy" << endl;
	  
	  cout << "start free" << endl;


	  hipFree(d_kernelCenter);
	  hipFree(d_step);
	  hipFree(d_kernelWidth);
	  hipFree(d_kernel);
	  
	  cout << "end free" << endl;
	  cout << kernelCenter[0] << endl;
	  cout << step[0] << endl;
	  cout << kernelWidth[0] << endl;
	  cout << kernel[0] << endl;
	  */
	return 0;
}
